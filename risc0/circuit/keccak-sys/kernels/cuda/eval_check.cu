#include "hip/hip_runtime.h"
// Copyright 2025 RISC Zero, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "eval_check.cuh"

#include "hip/hip_runtime.h"
#include "supra/fp.h"
#include <exception>
#include <iostream>

namespace risc0::circuit::keccak::cuda {

using MutableBuf = const Fp*;
using GlobalBuf = const Fp*;
using GlobalExtBuf = const FpExt*;
using ExtVal = FpExt;
using Val = Fp;
using Index = size_t;
using MixState = FpExt;

constexpr bool kDebug = false;
__constant__ FpExt poly_mix[kNumPolyMixPows];

constexpr size_t kInvRate = 4;

template <size_t N> __device__ __inline__ size_t readBits(const uint8_t* &bc, const char* label) {
  assert((N % 8) == 0);
  size_t bytes = N / 8;
  size_t result = 0;

  for (size_t i = 0; i != bytes; i++) {
    result += (*bc++) << (i * 8);
  }
  if (kDebug) {
    printf(" decoded %lu (%s)\n", result, label);
  }
  return result;
}

template <typename T, size_t N> __device__ __inline__ T& getFromTemp(T (&tempBuf)[N], size_t idx) {
  assert(idx < N);
  return tempBuf[idx];
}

#define zllGet(BUF, OFFSET, BACK) ((BUF)[(OFFSET) * steps + ((cycle - kInvRate * (BACK)) & mask)]);
#define zllGetGlobal(BUF, OFFSET) ((BUF)[(OFFSET)])
#define debugIn(X) (X)
#define debugOut(X)                                                                                \
  do {                                                                                             \
  } while (0)

__device__ __inline__ Fp zllConst(int a) {
  return Fp(a);
}

__device__ __inline__ FpExt zllConst(int a, int b, int c, int d) {
  return FpExt(a, b, c, d);
}

__device__ __inline__ FpExt trivialConstraint() {
  return FpExt(0, 0, 0, 0);
}

__device__ __inline__ FpExt zllAndEqz(FpExt inMix, Fp val, size_t mixPowIndex) {
  return inMix + val * poly_mix[mixPowIndex];
}

__device__ __inline__ FpExt zllAndEqz(FpExt inMix, FpExt val, size_t mixPowIndex) {
  return inMix + val * poly_mix[mixPowIndex];
}

__device__ __inline__ FpExt zllAndCond(FpExt inMix, Fp cond, FpExt innerMix, size_t mixPowIndex) {
  return inMix + cond * innerMix * poly_mix[mixPowIndex];
}

__device__ __inline__ FpExt
zllAndCond(FpExt inMix, FpExt cond, FpExt innerMix, size_t mixPowIndex) {
  return inMix + cond * innerMix * poly_mix[mixPowIndex];
}

#include "eval_check_bc.cu.inc"

__global__ void eval_check(Fp* check,
                           const Fp* ctrl,
                           const Fp* data,
                           const Fp* accum,
                           const Fp* mix,
                           const Fp* out,
                           const Fp rou,
                           uint32_t po2,
                           uint32_t domain) {
  uint32_t cudaCycle = blockDim.x * blockIdx.x + threadIdx.x;
  if (kDebug) {
    if (cudaCycle != 0)
      return;
  }
  for (uint32_t cycle = cudaCycle; cycle != domain; ++cudaCycle) {
    if (cycle < domain) {
      FpExt tot = keccak(cycle, domain, data, out, poly_mix);
      Fp x = pow(rou, cycle);
      Fp y = pow(Fp(3) * x, 1 << po2);
      FpExt ret = tot * inv(y - Fp(1));
      check[domain * 0 + cycle] = ret[0];
      check[domain * 1 + cycle] = ret[1];
      check[domain * 2 + cycle] = ret[2];
      check[domain * 3 + cycle] = ret[3];
    }
    if (!kDebug)
      break;
  }
}

} // namespace risc0::circuit::keccak::cuda

extern "C" {

using namespace risc0::circuit::keccak::cuda;

const char* risc0_circuit_keccak_cuda_eval_check(Fp* check,
                                                 const Fp* ctrl,
                                                 const Fp* data,
                                                 const Fp* accum,
                                                 const Fp* mix,
                                                 const Fp* out,
                                                 const Fp& rou,
                                                 uint32_t po2,
                                                 uint32_t domain,
                                                 const FpExt* poly_mix_pows) {
  try {
    CUDA_OK(hipDeviceSynchronize());
    CudaStream stream;
    auto cfg = getSimpleConfig(domain);
    hipMemcpyToSymbol(HIP_SYMBOL(poly_mix), poly_mix_pows, sizeof(poly_mix));
    eval_check<<<cfg.grid, cfg.block, 0, stream>>>(
        check, ctrl, data, accum, mix, out, rou, po2, domain);
    CUDA_OK(hipStreamSynchronize(stream));
  } catch (const std::exception& err) {
    return strdup(err.what());
  } catch (...) {
    return strdup("Generic exception");
  }
  return nullptr;
}

} // extern "C"
